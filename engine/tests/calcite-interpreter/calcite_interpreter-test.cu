#include <cstdlib>
#include <iostream>
#include <string>
#include <vector>

#include "gtest/gtest.h"
#include <CalciteExpressionParsing.h>
#include <CalciteInterpreter.h>
#include <DataFrame.h>
#include <GDFColumn.cuh>
#include <GDFCounter.cuh>
#include <Utils.cuh>
#include <blazingdb/io/Util/StringUtil.h>

#include "blazingdb/io/Library/Logging/ServiceLogging.h"
#include <blazingdb/io/Library/Logging/CoutOutput.h>
#include <blazingdb/io/Library/Logging/Logger.h>
#include "../BlazingUnitTest.h"

class TestEnvironment : public testing::Environment {
public:
  virtual ~TestEnvironment() {}
  virtual void SetUp() {
    auto output = new Library::Logging::CoutOutput();
    Library::Logging::ServiceLogging::getInstance().setLogOutput(output);
  }

  void TearDown() {
    hipDeviceReset(); // for cuda-memchecking
  }
};

struct calcite_interpreter_TEST : public BlazingUnitTest {

  void SetUp() {
	  
	input1 = new char[num_values];
    input2 = new char[num_values];
    input3 = new char[num_values];

    for (std::size_t i = 0; i < num_values; i++) {
      if (i % 2 == 0) {
        input1[i] = 1;
      } else {
        input1[i] = i;
      }
      input2[i] = i;
      input3[i] = 1;
    }

    inputs.resize(3);
	gdf_dtype_extra_info extra_info{TIME_UNIT_NONE};
    inputs[0].create_gdf_column(GDF_INT8, extra_info, num_values, (void *)input1, 1, "");
    inputs[1].create_gdf_column(GDF_INT8, extra_info, num_values, (void *)input2, 1, "");
    inputs[2].create_gdf_column(GDF_INT8, extra_info, num_values, (void *)input3, 1, "");

    input_tables.push_back(inputs); // columns for emps
    input_tables.push_back(inputs); // columns for sales
  }

  void TearDown() {
    // Releasing allocated memory, here we are responsible for that
    // TODO percy rommel: move to integration/end-to-end test
    // GDFRefCounter::getInstance()->free_if_deregistered(outputs[i].get_gdf_column())
  }

  void Check(gdf_column_cpp out_col, char *host_output) {

    char *device_output;
    device_output = new char[out_col.size()];
    hipMemcpy(device_output, out_col.data(), out_col.size() * WIDTH_PER_VALUE,
               hipMemcpyDeviceToHost);

    for (int i = 0; i < out_col.size(); i++) {
      // std::cout<<(int)host_output[i]<<" =?=
      // "<<(int)device_output[i]<<std::endl<<std::flush;
      EXPECT_TRUE(host_output[i] == device_output[i]);
    }
  }

  gdf_column_cpp left;
  gdf_column_cpp right;
  gdf_column_cpp third;

  std::vector<gdf_column_cpp> inputs;

  char *input1;
  char *input2;
  char *input3;

  size_t num_values = 32;

  std::vector<std::vector<gdf_column_cpp>> input_tables;
  std::vector<std::string> table_names = {"hr.emps", "hr.sales"};
  std::vector<std::vector<std::string>> column_names = {{"x", "y", "z"},
                                                        {"a", "b", "x"}};

  std::vector<gdf_column_cpp> outputs;

  const int WIDTH_PER_VALUE = 1;
};
 
TEST_F(calcite_interpreter_TEST, processing_project6) {

  { // select x - y as S from hr.emps
    std::string query = "\
LogicalProject(S=[-($0, $1)])\n\
  LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    char *host_output = new char[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      host_output[i] = input1[i] - input2[i];
    }

    print_gdf_column(outputs[0].get_gdf_column());
    Check(outputs[0], host_output);
  }
}

TEST_F(calcite_interpreter_TEST, order_by) {

  {
    size_t num_values = 32;

    uint64_t *data_test = new uint64_t[num_values];
    int *data = new int[num_values];
    for (size_t i = 0; i < num_values; i++) {
      data[i] = num_values - i;
      data_test[i] = i;
    }

    gdf_column_cpp input_column;

    gdf_column_cpp indices_col;

    std::cout << "now running other version" << std::endl;
	gdf_dtype_extra_info extra_info{TIME_UNIT_NONE};

    input_column.create_gdf_column(GDF_INT32, extra_info, num_values, (void *)data, 4, "");

    // TODO percy noboa see upgrade to uints
    // indices_col.create_gdf_column(GDF_UINT64,num_values,nullptr,8);
    indices_col.create_gdf_column(GDF_INT64, extra_info, num_values, nullptr, 8, "");
    print_gdf_column(indices_col.get_gdf_column());
    cudf::valid_type asc_desc_bitmask = 255;
    cudf::valid_type *asc_desc_bitmask_dev;

    hipMalloc((void **)&asc_desc_bitmask_dev, 1);

    hipError_t err2 = hipMemcpy(asc_desc_bitmask_dev, &asc_desc_bitmask, 1,
                                  hipMemcpyHostToDevice);

    std::vector<gdf_column> v_cols(1);
    for (auto i = 0; i < 1; ++i) {
      v_cols[i] = *(input_column.get_gdf_column());
    }

    gdf_column *input_columns = &v_cols[0];

    print_gdf_column(input_column.get_gdf_column());
    try {
      // TODO percy noboa felipe see upgrade to order_by
      //		    gdf_error err = gdf_order_by_asc_desc(
      //					input_columns,
      //					1,
      //					indices_col.get_gdf_column(),
      //					asc_desc_bitmask_dev);
      //			EXPECT_TRUE(err == GDF_SUCCESS);

    } catch (std::exception e) {

      std::cout << "We caught an exception running order by!" << e.what()
                << std::endl;
    }
    std::cout << "printing size " << indices_col.size() << std::endl;
    print_typed_column<uint64_t>((uint64_t *)indices_col.get_gdf_column()->data,
                                 nullptr, indices_col.size());
    delete[] data;
    hipFree(asc_desc_bitmask_dev);
 
  }
}

TEST_F(calcite_interpreter_TEST, processing_sort) {

  { // select x - y as S from hr.emps
    std::string query = "LogicalSort(sort0=[$0], dir0=[ASC])\n\
  LogicalProject(x=[$0], x=[$1])\n\
    LogicalTableScan(table=[[hr, emps]])";
    std::cout << "about to evalute" << std::endl;
    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    std::cout << "evaluated" << std::endl;
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 2);

    for (std::size_t i = 0; i < outputs.size(); i++) {
      print_gdf_column(outputs[i].get_gdf_column());
    }

    std::vector<char> output = {1,  1,  1,  1,  1,  1,  1,  1,  1,  1, 1,
                                1,  1,  1,  1,  1,  1,  3,  5,  7,  9, 11,
                                13, 15, 17, 19, 21, 23, 25, 27, 29, 31};
    char *host_output = &output[0];
    Check(outputs[0], host_output);
    output = {0,  1, 2, 4, 6, 8,  10, 12, 14, 16, 18, 20, 22, 24, 26, 28,
              30, 3, 5, 7, 9, 11, 13, 15, 17, 19, 21, 23, 25, 27, 29, 31};
    host_output = &output[0];
    Check(outputs[1], host_output);
  }
}

int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  ::testing::Environment *const env =
      ::testing::AddGlobalTestEnvironment(new TestEnvironment());
  return RUN_ALL_TESTS();
}
