#include <cstdlib>
#include <iostream>
#include <string>
#include <vector>

#include "gtest/gtest.h"
#include <CalciteExpressionParsing.h>
#include <CalciteInterpreter.h>
#include <DataFrame.h>
#include <GDFColumn.cuh>
#include <GDFCounter.cuh>
#include <Utils.cuh>
#include <blazingdb/io/Util/StringUtil.h>
#include <gdf/library/table_group.h>
#include "../BlazingUnitTest.h"


class TestEnvironment : public testing::Environment {
public:
  virtual ~TestEnvironment() {}
  virtual void SetUp() {  }

  void TearDown() {
    hipDeviceReset(); // for cuda-memchecking
  }
};

using gdf::library::DType;
using gdf::library::Index;
using gdf::library::TableGroupBuilder;

using RType = DType<GDF_INT32>;

struct calcite_interpreter_TEST : public  BlazingUnitTest {

  gdf::library::TableGroup group;

  calcite_interpreter_TEST()
      : group{TableGroupBuilder{
            {"hr.emps",
             {
                 {"x", [](Index i) -> RType { return i % 2 ? i : 1; }},
                 {"y", [](Index i) -> RType { return i; }},
                 {"z", [](Index) -> RType { return 1; }},
             }},
        }
                  .Build(num_values)} {}

  void SetUp() {
	  input_tables = group.ToBlazingFrame();

    input1 = reinterpret_cast<const std::int32_t *>(group[0][0].get(0));
    input2 = reinterpret_cast<const std::int32_t *>(group[0][1].get(0));
    input3 = reinterpret_cast<const std::int32_t *>(group[0][2].get(0));
  }

  void TearDown() {

    for (std::size_t i = 0; i < outputs.size(); i++) {

      // Releasing allocated memory, here we are responsible for that
      // TODO percy rommel: move to integration/end-to-end test
      // GDFRefCounter::getInstance()->free_if_deregistered(outputs[i].get_gdf_column());
    }
  }

  void Check(gdf_column_cpp out_col, int32_t *host_output,
             size_t num_output_values = 0) {

    if (num_output_values == 0)
      num_output_values = out_col.size();

    int32_t *device_output;
    device_output = new int32_t[num_output_values];
    hipMemcpy(device_output, out_col.data(),
               num_output_values * sizeof(int32_t), hipMemcpyDeviceToHost);

    for (std::size_t i = 0; i < num_output_values; i++) {
      EXPECT_TRUE(host_output[i] == device_output[i]);
    }
  }

  gdf_column_cpp left;
  gdf_column_cpp right;
  gdf_column_cpp third;

  std::vector<gdf_column_cpp> inputs;

  static const std::size_t num_values = 32;

  const std::int32_t *input1;
  const std::int32_t *input2;
  const std::int32_t *input3;

  std::vector<std::vector<gdf_column_cpp>> input_tables;
  std::vector<std::string> table_names = {"hr.emps"};
  std::vector<std::vector<std::string>> column_names = {{"x", "y", "z"}};

  std::vector<gdf_column_cpp> outputs;
};

TEST_F(calcite_interpreter_TEST, where_greater) {

  { // select y from hr.emps where y > 5
    std::string query = "\
LogicalProject(y=[$1])\n\
  LogicalFilter(condition=[>($1, 5)])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int cur = 0;
    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      if (input2[i] > 5) {
        host_output[cur] = input2[i];
        cur++;
      }
    }

    Check(outputs[0], host_output, cur);
  }
}

TEST_F(calcite_interpreter_TEST, where_equals) {

  { // select z from hr.emps where x = y
    std::string query = "\
LogicalProject(z=[$2])\n\
  LogicalFilter(condition=[=($0, $1)])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int cur = 0;
    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      if (input1[i] == input2[i]) {
        host_output[cur] = input3[i];
        cur++;
      }
    }

    Check(outputs[0], host_output, cur);
  }
}

// ToDo: fix both literals returns invalid_api_call
TEST_F(calcite_interpreter_TEST, DISABLED_processing_project51) {

  { // select z from hr.emps where 1 = 1
    std::string query = "\
LogicalProject(z=[$2])\n\
  LogicalFilter(condition=[=(1, 1)])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      host_output[i] = input3[i];
    }

    Check(outputs[0], host_output);
  }
}

TEST_F(calcite_interpreter_TEST, where_withmod) {

  { // select y from hr.emps where mod(y,5)=0"
    std::string query = "\
LogicalProject(y=[$1])\n\
  LogicalFilter(condition=[=(MOD($1, 5), 0)])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int cur = 0;
    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      if ((input2[i]) % 5 == 0) {
        host_output[cur] = input2[i];
        cur++;
      }
    }

    Check(outputs[0], host_output, cur);
  }
}

TEST_F(calcite_interpreter_TEST, where_equals_and_greater) {

  { // select z from hr.emps where x = y and x > 4
    std::string query = "\
LogicalProject(z=[$2])\n\
  LogicalFilter(condition=[AND(=($1, $0), >($0, 4))])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int cur = 0;
    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      if (input1[i] == input2[i] && input1[i] > 4) {
        host_output[cur] = input3[i];
        cur++;
      }
    }

    Check(outputs[0], host_output, cur);
  }
}

TEST_F(calcite_interpreter_TEST, where_not_equals) {

  { // select x from hr.emps where x <> 1
    std::string query = "\
LogicalProject(x=[$0])\n\
  LogicalFilter(condition=[<>($0, 1)])\n\
    LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 1);

    int cur = 0;
    int32_t *host_output = new int32_t[num_values];
    for (std::size_t i = 0; i < num_values; i++) {
      if (input1[i] != 1) {
        host_output[cur] = input1[i];
        cur++;
      }
    }

    Check(outputs[0], host_output, cur);
  }
}

// Output with several columns (all of them)
TEST_F(calcite_interpreter_TEST, where_all_columns_withand) {

  { // select * from hr.emps where x = z AND 4 >= y
    std::string query = "\
LogicalProject(x=[$0], y=[$1], z=[$2])\n\
  LogicalFilter(condition=[AND(=($0, $2), >=(4, $1))])\n\
	LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 3);

    int cur = 0;
    int32_t *host_output1 = new int32_t[num_values];
    int32_t *host_output2 = new int32_t[num_values];
    int32_t *host_output3 = new int32_t[num_values];

    for (std::size_t i = 0; i < num_values; i++) {
      if (input1[i] == input3[i] && 4 >= input2[i]) {
        host_output1[cur] = input1[i];
        host_output2[cur] = input2[i];
        host_output3[cur] = input3[i];
        cur++;
      }
    }

    Check(outputs[0], host_output1, cur);
    Check(outputs[1], host_output2, cur);
    Check(outputs[2], host_output3, cur);
  }
}

// Output with several columns (two of them)
TEST_F(calcite_interpreter_TEST, where_two_columns_withand) {

  { // select x,z from hr.emps where x = 1 AND 4 >= y
    std::string query = "\
LogicalProject(x=[$0], z=[$2])\n\
  LogicalFilter(condition=[AND(=(1, $0), >=(4, $1))])\n\
	LogicalTableScan(table=[[hr, emps]])";

    gdf_error err =
        evaluate_query(input_tables, table_names, column_names, query, outputs);
    EXPECT_TRUE(err == GDF_SUCCESS);
    EXPECT_TRUE(outputs.size() == 2);

    int cur = 0;
    int32_t *host_output1 = new int32_t[num_values];
    int32_t *host_output2 = new int32_t[num_values];

    for (std::size_t i = 0; i < num_values; i++) {
      if (input1[i] == 1 && 4 >= input2[i]) {
        host_output1[cur] = input1[i];
        host_output2[cur] = input3[i];
        cur++;
      }
    }

    Check(outputs[0], host_output1, cur);
    Check(outputs[1], host_output2, cur);
  }
}
 