/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "column_utilities.hpp"

#include <cudf/column/column_view.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/bit.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/detail/copy.hpp>

#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/column_wrapper.hpp>

#include <thrust/equal.h>

#include <gmock/gmock.h>

namespace cudf {
namespace test {

// Property equality
void expect_column_properties_equal(cudf::column_view const& lhs, cudf::column_view const& rhs) {
  EXPECT_EQ(lhs.type(), rhs.type());
  EXPECT_EQ(lhs.size(), rhs.size());
  EXPECT_EQ(lhs.null_count(), rhs.null_count());
  if (lhs.size() > 0) {
     EXPECT_EQ(lhs.nullable(), rhs.nullable());
  }
  EXPECT_EQ(lhs.has_nulls(), rhs.has_nulls());
  EXPECT_EQ(lhs.num_children(), rhs.num_children());
}

class corresponding_rows_unequal {
public:
  corresponding_rows_unequal(table_device_view d_lhs, table_device_view d_rhs): comp(d_lhs, d_rhs) {
  }
  
  cudf::experimental::row_equality_comparator<true> comp;
    
  __device__ bool operator()(size_type index) {
    return !comp(index, index);
  }
};

void expect_columns_equal(cudf::column_view const& lhs, cudf::column_view const& rhs,
                          bool print_all_differences) {
  expect_column_properties_equal(lhs, rhs);

  auto d_lhs = cudf::table_device_view::create(table_view{{lhs}});
  auto d_rhs = cudf::table_device_view::create(table_view{{rhs}});

  thrust::device_vector<int> differences(lhs.size());

  auto diff_iter = thrust::copy_if(thrust::device,
                                   thrust::make_counting_iterator(0),
                                   thrust::make_counting_iterator(lhs.size()),
                                   differences.begin(),
                                   corresponding_rows_unequal(*d_lhs, *d_rhs));

  CUDA_TRY(hipDeviceSynchronize());

  differences.resize(thrust::distance(differences.begin(), diff_iter));

  if (diff_iter > differences.begin()) {
    if (print_all_differences) {
      //
      //  If there are differences, display them all
      //
      std::ostringstream buffer;
      buffer << "differences:" << std::endl;
      
      cudf::table_view source_table ({lhs, rhs});

      fixed_width_column_wrapper<int32_t> diff_column(differences.begin(), differences.end());

      std::unique_ptr<cudf::experimental::table> diff_table = cudf::experimental::gather(source_table,
											 diff_column);
      
      //
      //  Need to pull back the differences
      //
      std::vector<std::string> h_left_strings = to_strings(diff_table->get_column(0));
      std::vector<std::string> h_right_strings = to_strings(diff_table->get_column(1));

      for (size_t i = 0 ; i < differences.size() ; ++i) {
          buffer << "lhs[" << differences[i] << "] = " << h_left_strings[i]
                 << ", rhs[" << differences[i] << "] = " << h_right_strings[i] << std::endl;
      }

      EXPECT_EQ(differences.size(), size_t{0}) << buffer.str();
    } else {
      //
      //  If there are differences, just display the first one
      //
      int index = differences[0];

      auto diff_lhs = cudf::experimental::detail::slice(lhs, index, index+1);
      auto diff_rhs = cudf::experimental::detail::slice(rhs, index, index+1);

      std::vector<std::string> h_left_strings = to_strings(diff_lhs);
      std::vector<std::string> h_right_strings = to_strings(diff_rhs);

      EXPECT_EQ(differences.size(), size_t{0}) << "first difference: "
                                               << "lhs[" << index << "] = "
                                               << to_string(diff_lhs, "")
                                               << ", rhs[" << index << "] = "
                                               << to_string(diff_rhs, "");
    }
  }
}

// Bitwise equality
void expect_equal_buffers(void const* lhs, void const* rhs,
                          std::size_t size_bytes) {
  if (size_bytes > 0) {
    EXPECT_NE(nullptr, lhs);
    EXPECT_NE(nullptr, rhs);
  }
  auto typed_lhs = static_cast<char const*>(lhs);
  auto typed_rhs = static_cast<char const*>(rhs);
  EXPECT_TRUE(thrust::equal(thrust::device, typed_lhs, typed_lhs + size_bytes,
                            typed_rhs));
}

// copy column bitmask to host (used by to_host())
std::vector<bitmask_type> bitmask_to_host(cudf::column_view const& c) {
  if (c.nullable()) {
    auto num_bitmasks = bitmask_allocation_size_bytes(c.size()) / sizeof(bitmask_type);
    std::vector<bitmask_type> host_bitmask(num_bitmasks);

    CUDA_TRY(hipMemcpy(host_bitmask.data(), c.null_mask(), num_bitmasks * sizeof(bitmask_type),
                        hipMemcpyDeviceToHost));

    return host_bitmask;
  }
  else {
    return std::vector<bitmask_type>{};
  }
}


struct column_view_printer {
  template <typename Element, typename std::enable_if_t<is_numeric<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string> & out) {
    auto h_data = cudf::test::to_host<Element>(col);

    out.resize(col.size());

    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx) ? std::to_string(h_data.first[idx]) : std::string("NULL");
                     });
    } else {
      std::transform(h_data.first.begin(), h_data.first.end(), out.begin(), [](Element el) {
          return std::to_string(el);
        });
    }
  }

  template <typename Element, typename std::enable_if_t<is_timestamp<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string> & out) {
    //
    //  For timestamps, convert timestamp column to column of strings, then
    //  call string version
    //
    auto col_as_strings = cudf::strings::from_timestamps(col);

    this->template operator()<cudf::string_view>(*col_as_strings, out);
  }

  template <typename Element, typename std::enable_if_t<std::is_same<Element, cudf::string_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string> & out) {
    //
    //  Implementation for strings, call special to_host variant
    //
    auto h_data = cudf::test::to_host<std::string>(col);

    out.resize(col.size());
    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx) ? h_data.first[idx] : std::string("NULL");
                     });
    } else {
      out = std::move(h_data.first);
    }
  }
};

std::vector<std::string> to_strings(cudf::column_view const& col) {
  std::vector<std::string> reply;

  cudf::experimental::type_dispatcher(col.type(),
                                      column_view_printer{}, 
                                      col,
                                      reply);

  return reply;
}

std::string to_string(cudf::column_view const& col, std::string const& delimiter) {

  std::ostringstream buffer;
  std::vector<std::string> h_data = to_strings(col);

  std::copy(h_data.begin(), h_data.end() - 1, std::ostream_iterator<std::string>(buffer, delimiter.c_str()));
  buffer << h_data.back();

  return buffer.str();
}

void print(cudf::column_view const& col, std::ostream &os, std::string const& delimiter) {
  os << to_string(col, delimiter);
}

}  // namespace test
}  // namespace cudf
