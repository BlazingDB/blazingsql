#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"

namespace ral {
namespace config {

size_t gpuMemorySize() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CheckCudaErrors( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

}	// namespace config
}	// namespace ral
