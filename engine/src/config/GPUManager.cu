#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudf/utilities/error.hpp>
#include "GPUManager.cuh"

namespace ral {
namespace config {

size_t gpuFreeMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

size_t gpuTotalMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total;
}

size_t gpuUsedMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total - free;
}

}	// namespace config
}	// namespace ral
