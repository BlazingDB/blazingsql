#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <exception>
#include "GPUManager.cuh"
#include "Utils.cuh"


namespace ral {
namespace config {

GPUManager::GPUManager() : currentDeviceId{0} {
	CheckCudaErrors( hipGetDeviceCount(&totalDevices) );
}

int GPUManager::getDeviceId() {
	return this->currentDeviceId;
}


GPUManager& GPUManager::getInstance() {
	static GPUManager instance;
	return instance;
}

void GPUManager::initialize(int deviceId) {
	if (deviceId < 0 || deviceId >= totalDevices) {
		throw std::runtime_error("In GPUManager::initialize function: Invalid deviceId");
	}

	currentDeviceId = deviceId;
	setDevice();
}

void GPUManager::setDevice() {
	CheckCudaErrors( hipSetDevice(currentDeviceId) );
}

size_t GPUManager::gpuMemorySize() {
	struct hipDeviceProp_t props;
	CheckCudaErrors( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

}	// namespace config
}	// namespace ral
