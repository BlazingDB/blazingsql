
#include "MessageUtil.cuh"

#include <cudf/copying.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <thrust/transform.h>

#include <from_cudf/cpp_tests/utilities/column_utilities.hpp>


namespace ral {
namespace communication {
namespace messages {
namespace experimental {

	std::pair<cudf::size_type, cudf::size_type> getCharsColumnStartAndEnd(const CudfColumnView & column){
		
		cudf::size_type offset = column.offset();
		CudfColumnView offsets_column = column.child(0);

		cudf::size_type chars_column_start, chars_column_end;
		hipMemcpy(&chars_column_start, (void*)(offsets_column.begin<cudf::size_type>() + offset), sizeof(cudf::size_type), hipMemcpyDeviceToHost);
		hipMemcpy(&chars_column_end, (void*)(offsets_column.begin<cudf::size_type>() + offset + column.size()), sizeof(cudf::size_type), hipMemcpyDeviceToHost);
		return std::make_pair(chars_column_start, chars_column_end);
	}
	
	std::unique_ptr<CudfColumn> getRebasedStringOffsets(const CudfColumnView & column, cudf::size_type chars_column_start){

		CudfColumnView offsets_column = column.child(0);
	
		// NOTE that the offsets column size is usually one more than the number of strings. It starts at 0 and ends at chars_column.size()
		cudf::size_type offset = column.offset();
		std::unique_ptr<CudfColumn> new_offsets = cudf::experimental::allocate_like(offsets_column, 
				column.size() + 1, cudf::experimental::mask_allocation_policy::NEVER);

		auto mutable_col = new_offsets->mutable_view();

		cudf::experimental::copy_range(offsets_column, mutable_col,
				offset, offset + column.size() + 1, 0);

		struct subtracting_operator	{
			cudf::size_type _sub;
			subtracting_operator(cudf::size_type sub){
				_sub = sub;
			}
			__host__ __device__
			cudf::size_type operator()(const cudf::size_type x)
			{
				return (x - _sub);
			}
		};
	
		subtracting_operator op(chars_column_start);
		auto start_src = mutable_col.begin<cudf::size_type>();
		auto end_src = mutable_col.end<cudf::size_type>();
		auto start_dst = mutable_col.begin<cudf::size_type>();		
		thrust::transform(start_src, end_src, start_dst, op);

		return new_offsets;
	}

}  // namespace experimental
}  // namespace messages
}  // namespace communication
}  // namespace ral